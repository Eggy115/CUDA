#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011-2022, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include <cub/block/block_radix_sort.cuh>

#include <algorithm>
#include <type_traits>
#include <utility>

#include <thrust/host_vector.h>
#include <thrust/sequence.h>

// Has to go after all cub headers. Otherwise, this test won't catch unused
// variables in cub kernels.
#include "catch2_test_helper.h"

template <typename InputIteratorT,
          typename OutputIteratorT,
          typename ActionT,
          int ItemsPerThread,
          int ThreadsInBlock,
          int RadixBits,
          bool Memoize,
          hipcub::BlockScanAlgorithm Algorithm,
          hipSharedMemConfig ShmemConfig>
__global__ void kernel(
    ActionT action, 
    InputIteratorT input, 
    OutputIteratorT output,
    int begin_bit,
    int end_bit,
    bool striped)
{
  using key_t = hipcub::detail::value_t<InputIteratorT>;
  using block_radix_sort_t = hipcub::BlockRadixSort<key_t,
                                                 ThreadsInBlock,
                                                 ItemsPerThread,
                                                 hipcub::NullType,
                                                 RadixBits,
                                                 Memoize,
                                                 Algorithm,
                                                 ShmemConfig>;

  using storage_t = typename block_radix_sort_t::TempStorage;

  __shared__ storage_t storage;

  key_t keys[ItemsPerThread];

  for (int i = 0; i < ItemsPerThread; i++)
  {
    keys[i] = input[threadIdx.x * ItemsPerThread + i];
  }

  block_radix_sort_t block_radix_sort(storage);

  if (striped)
  {
    action(block_radix_sort,
           keys,
           begin_bit,
           end_bit,
           hipcub::Int2Type<1>{});

    for (int i = 0; i < ItemsPerThread; i++)
    {
      output[threadIdx.x + ThreadsInBlock * i] = keys[i];
    }
  }
  else
  {
    action(block_radix_sort,
           keys,
           begin_bit,
           end_bit,
           hipcub::Int2Type<0>{});

    for (int i = 0; i < ItemsPerThread; i++)
    {
      output[threadIdx.x * ItemsPerThread + i] = keys[i];
    }
  }
}

template <int ItemsPerThread,
          int ThreadsInBlock,
          int RadixBits,
          bool Memoize,
          hipcub::BlockScanAlgorithm Algorithm,
          hipSharedMemConfig ShmemConfig,
          typename InputIteratorT,
          typename OutputIteratorT,
          typename ActionT>
void block_radix_sort(
    ActionT action,
    InputIteratorT input,
    OutputIteratorT output,
    int begin_bit,
    int end_bit,
    bool striped)
{
  hipDeviceSetSharedMemConfig(ShmemConfig);

  kernel<InputIteratorT,
         OutputIteratorT,
         ActionT,
         ItemsPerThread,
         ThreadsInBlock,
         RadixBits,
         Memoize,
         Algorithm,
         ShmemConfig>
    <<<1, ThreadsInBlock>>>(action, input, output, begin_bit, end_bit, striped);

  REQUIRE( hipSuccess == hipPeekAtLastError() );
  REQUIRE( hipSuccess == hipDeviceSynchronize() );
}

template <typename InputKeyIteratorT,
          typename InputValueIteratorT,
          typename OutputKeyIteratorT,
          typename OutputValueIteratorT,
          typename ActionT,
          int ItemsPerThread,
          int ThreadsInBlock,
          int RadixBits,
          bool Memoize,
          hipcub::BlockScanAlgorithm Algorithm,
          hipSharedMemConfig ShmemConfig>
__global__ void kernel(
    ActionT action, 
    InputKeyIteratorT input_keys, 
    InputValueIteratorT input_values,
    OutputKeyIteratorT output_keys, 
    OutputValueIteratorT output_values,
    int begin_bit,
    int end_bit,
    bool striped)
{
  using key_t = hipcub::detail::value_t<InputKeyIteratorT>;
  using value_t = hipcub::detail::value_t<InputValueIteratorT>;
  using block_radix_sort_t = hipcub::BlockRadixSort<key_t,
                                                 ThreadsInBlock,
                                                 ItemsPerThread,
                                                 value_t,
                                                 RadixBits,
                                                 Memoize,
                                                 Algorithm,
                                                 ShmemConfig>;

  using storage_t = typename block_radix_sort_t::TempStorage;
  __shared__ storage_t storage;

  key_t keys[ItemsPerThread];
  value_t values[ItemsPerThread];

  for (int i = 0; i < ItemsPerThread; i++)
  {
    keys[i] = input_keys[threadIdx.x * ItemsPerThread + i];
    values[i] = input_values[threadIdx.x * ItemsPerThread + i];
  }

  block_radix_sort_t block_radix_sort(storage);

  if (striped)
  {
    action(block_radix_sort,
           keys,
           values,
           begin_bit,
           end_bit,
           hipcub::Int2Type<1>{});

    for (int i = 0; i < ItemsPerThread; i++)
    {
      output_keys[threadIdx.x + ThreadsInBlock * i] = keys[i];
      output_values[threadIdx.x + ThreadsInBlock * i] = values[i];
    }
  }
  else 
  {
    action(block_radix_sort,
           keys,
           values,
           begin_bit,
           end_bit,
           hipcub::Int2Type<0>{});

    for (int i = 0; i < ItemsPerThread; i++)
    {
      output_keys[threadIdx.x * ItemsPerThread + i] = keys[i];
      output_values[threadIdx.x * ItemsPerThread + i] = values[i];
    }
  }
}

template <int ItemsPerThread,
          int ThreadsInBlock,
          int RadixBits,
          bool Memoize,
          hipcub::BlockScanAlgorithm Algorithm,
          hipSharedMemConfig ShmemConfig,
          typename InputKeyIteratorT,
          typename InputValueIteratorT,
          typename OutputKeyIteratorT,
          typename OutputValueIteratorT,
          typename ActionT>
void block_radix_sort(
    ActionT action,
    InputKeyIteratorT input_keys,
    InputValueIteratorT input_values,
    OutputKeyIteratorT output_keys,
    OutputValueIteratorT output_values,
    int begin_bit,
    int end_bit,
    bool striped)
{
  hipDeviceSetSharedMemConfig(ShmemConfig);

  kernel<InputKeyIteratorT,
         InputValueIteratorT,
         OutputKeyIteratorT,
         OutputValueIteratorT,
         ActionT,
         ItemsPerThread,
         ThreadsInBlock,
         RadixBits,
         Memoize,
         Algorithm,
         ShmemConfig><<<1, ThreadsInBlock>>>(action,
                                             input_keys,
                                             input_values,
                                             output_keys,
                                             output_values,
                                             begin_bit,
                                             end_bit,
                                             striped);

  REQUIRE( hipSuccess == hipPeekAtLastError() );
  REQUIRE( hipSuccess == hipDeviceSynchronize() );
}

struct sort_op_t
{
  template <class BlockRadixSortT, class KeysT>
  __device__ void operator()(BlockRadixSortT &block_radix_sort,
                             KeysT &keys,
                             int begin_bit,
                             int end_bit,
                             hipcub::Int2Type<0> /* striped */)
  {
    block_radix_sort.Sort(keys, begin_bit, end_bit);
  }

  template <class BlockRadixSortT, class KeysT>
  __device__ void operator()(BlockRadixSortT &block_radix_sort,
                             KeysT &keys,
                             int begin_bit,
                             int end_bit,
                             hipcub::Int2Type<1> /* striped */)
  {
    block_radix_sort.SortBlockedToStriped(keys, begin_bit, end_bit);
  }
};

struct descending_sort_op_t
{
  template <class BlockRadixSortT, class KeysT>
  __device__ void operator()(BlockRadixSortT &block_radix_sort,
                             KeysT &keys,
                             int begin_bit,
                             int end_bit,
                             hipcub::Int2Type<0> /* striped */)
  {
    block_radix_sort.SortDescending(keys, begin_bit, end_bit);
  }

  template <class BlockRadixSortT, class KeysT>
  __device__ void operator()(BlockRadixSortT &block_radix_sort,
                             KeysT &keys,
                             int begin_bit,
                             int end_bit,
                             hipcub::Int2Type<1> /* striped */)
  {
    block_radix_sort.SortDescendingBlockedToStriped(keys, begin_bit, end_bit);
  }
};

struct sort_pairs_op_t
{
  template <class BlockRadixSortT, class KeysT, class ValuesT>
  __device__ void operator()(BlockRadixSortT &block_radix_sort,
                             KeysT &keys,
                             ValuesT &values,
                             int begin_bit,
                             int end_bit,
                             hipcub::Int2Type<0> /* striped */)
  {
    block_radix_sort.Sort(keys, values, begin_bit, end_bit);
  }

  template <class BlockRadixSortT, class KeysT, class ValuesT>
  __device__ void operator()(BlockRadixSortT &block_radix_sort,
                             KeysT &keys,
                             ValuesT &values,
                             int begin_bit,
                             int end_bit,
                             hipcub::Int2Type<1> /* striped */)
  {
    block_radix_sort.SortBlockedToStriped(keys, values, begin_bit, end_bit);
  }
};

struct descending_sort_pairs_op_t
{
  template <class BlockRadixSortT, class KeysT, class ValuesT>
  __device__ void operator()(BlockRadixSortT &block_radix_sort,
                             KeysT &keys,
                             ValuesT &values,
                             int begin_bit,
                             int end_bit,
                             hipcub::Int2Type<0> /* striped */)
  {
    block_radix_sort.SortDescending(keys, values, begin_bit, end_bit);
  }

  template <class BlockRadixSortT, class KeysT, class ValuesT>
  __device__ void operator()(BlockRadixSortT &block_radix_sort,
                             KeysT &keys,
                             ValuesT &values,
                             int begin_bit,
                             int end_bit,
                             hipcub::Int2Type<1> /* striped */)
  {
    block_radix_sort.SortDescendingBlockedToStriped(keys,
                                                    values,
                                                    begin_bit,
                                                    end_bit);
  }
};

// %PARAM% TEST_MEMOIZE mem 0:1
// %PARAM% TEST_ALGORITHM alg 0:1
// %PARAM% TEST_IPT ipt 1:11
// %PARAM% TEST_THREADS_IN_BLOCK ipt 32:160

using types = c2h::type_list<std::uint8_t, std::uint16_t, std::uint32_t, std::uint64_t>;
using no_value_types = c2h::type_list<hipcub::NullType>;

using key_types = c2h::type_list<std::int8_t, std::int16_t, std::int32_t, std::int64_t, float, double>;
using value_types = c2h::type_list<std::int8_t, c2h::custom_type_t<c2h::equal_comparable_t>>;

using threads_in_block = c2h::enum_type_list<int, TEST_THREADS_IN_BLOCK>;
using items_per_thread = c2h::enum_type_list<int, TEST_IPT>;
using radix_bits = c2h::enum_type_list<int, 1, 5>;
using memoize = c2h::enum_type_list<bool, TEST_MEMOIZE>;

#if TEST_ALGORITHM == 0
using algorithm = c2h::enum_type_list<hipcub::BlockScanAlgorithm, 
      hipcub::BlockScanAlgorithm::BLOCK_SCAN_RAKING>;
#else
using algorithm = c2h::enum_type_list<hipcub::BlockScanAlgorithm, 
      hipcub::BlockScanAlgorithm::BLOCK_SCAN_WARP_SCANS>; 
#endif

using shmem_config = c2h::enum_type_list<hipSharedMemConfig,
                                         hipSharedMemBankSizeFourByte,
                                         hipSharedMemBankSizeEightByte>;

using shmem_config_4 =
  c2h::enum_type_list<hipSharedMemConfig, hipSharedMemBankSizeFourByte>;

template <class TestType>
struct params_t
{
  using key_type = typename c2h::get<0, TestType>;
  using value_type = typename c2h::get<1, TestType>;

  static constexpr int items_per_thread = c2h::get<2, TestType>::value;
  static constexpr int threads_in_block = c2h::get<3, TestType>::value;
  static constexpr int tile_size = items_per_thread * threads_in_block;
  static constexpr int radix_bits = c2h::get<4, TestType>::value;
  static constexpr bool memoize = c2h::get<5, TestType>::value;
  static constexpr hipcub::BlockScanAlgorithm algorithm = 
    c2h::get<6, TestType>::value;
  static constexpr hipSharedMemConfig shmem_config = 
    c2h::get<7, TestType>::value;
};

template <class KeyT>
thrust::host_vector<KeyT>
get_striped_keys(const thrust::host_vector<KeyT> &h_keys,
                 int begin_bit,
                 int end_bit)
{
  thrust::host_vector<KeyT> h_striped_keys(h_keys);
  KeyT *h_striped_keys_data = thrust::raw_pointer_cast(h_striped_keys.data());

  if ((begin_bit > 0) || (end_bit < static_cast<int>(sizeof(KeyT) * 8)))
  {
    const int num_bits = end_bit - begin_bit;

    for (std::size_t i = 0; i < h_keys.size(); i++)
    {
      unsigned long long base = 0;
      memcpy(&base, h_striped_keys_data + i, sizeof(KeyT));
      base &= ((1ULL << num_bits) - 1) << begin_bit;
      memcpy(h_striped_keys_data + i, &base, sizeof(KeyT));
    }
  }

  return h_striped_keys;
}

template <class KeyT>
thrust::host_vector<std::size_t>
get_permutation(const thrust::host_vector<KeyT> &h_keys,
                bool is_descending,
                int begin_bit,
                int end_bit)
{
  thrust::host_vector<KeyT> h_striped_keys =
    get_striped_keys(h_keys, begin_bit, end_bit);

  thrust::host_vector<std::size_t> h_permutation(h_keys.size());
  thrust::sequence(h_permutation.begin(), h_permutation.end());

  std::stable_sort(h_permutation.begin(),
                   h_permutation.end(),
                   [&](std::size_t a, std::size_t b) {
                     if (is_descending)
                     {
                       return h_striped_keys[a] > h_striped_keys[b];
                     }

                     return h_striped_keys[a] < h_striped_keys[b];
                   });

  return h_permutation;
}

template <class KeyT>
thrust::host_vector<KeyT>
compute_reference(const thrust::device_vector<KeyT> &d_keys,
                  bool is_descending,
                  int begin_bit,
                  int end_bit)
{
  thrust::host_vector<KeyT> h_keys(d_keys);
  thrust::host_vector<std::size_t> h_permutation =
    get_permutation(h_keys, is_descending, begin_bit, end_bit);

  thrust::host_vector<KeyT> result(d_keys.size());
  std::transform(h_permutation.begin(),
                 h_permutation.end(),
                 result.begin(),
                 [&](std::size_t i) { return h_keys[i]; });

  return result;
}

template <class KeyT, class ValueT>
std::pair<thrust::host_vector<KeyT>, thrust::host_vector<ValueT>>
compute_reference(const thrust::device_vector<KeyT> &d_keys,
                  const thrust::device_vector<ValueT> &d_values,
                  bool is_descending,
                  int begin_bit,
                  int end_bit)
{
  std::pair<thrust::host_vector<KeyT>, thrust::host_vector<ValueT>> result;
  result.first.resize(d_keys.size());
  result.second.resize(d_keys.size());

  thrust::host_vector<KeyT> h_keys(d_keys);
  thrust::host_vector<std::size_t> h_permutation = get_permutation(h_keys, is_descending, begin_bit, end_bit);

  std::transform(h_permutation.begin(),
                 h_permutation.end(),
                 result.first.begin(),
                 [&](std::size_t i) { return h_keys[i]; });

  thrust::host_vector<ValueT> h_values(d_values);
  std::transform(h_permutation.begin(),
                 h_permutation.end(),
                 result.second.begin(),
                 [&](std::size_t i) { return h_values[i]; });

  return result;
}

CUB_TEST("Block radix sort can sort keys",
         "[radix][sort][block]",
         types,
         no_value_types,
         items_per_thread,
         threads_in_block,
         radix_bits,
         memoize,
         algorithm,
         shmem_config)
{
  using params = params_t<TestType>;
  using type = typename params::key_type;

  thrust::device_vector<type> d_output(params::tile_size);
  thrust::device_vector<type> d_input(params::tile_size);
  c2h::gen(CUB_SEED(2), d_input);

  const int key_size  = static_cast<int>(sizeof(type) * 8);
  const int begin_bit = GENERATE_COPY(take(2, random(0, key_size)));
  const int end_bit   = GENERATE_COPY(take(2, random(begin_bit, key_size)));
  const bool striped  = GENERATE_COPY(false, true);

  const bool is_descending = false;

  block_radix_sort<params::items_per_thread,
                   params::threads_in_block,
                   params::radix_bits,
                   params::memoize,
                   params::algorithm,
                   params::shmem_config>(
    sort_op_t{},
    thrust::raw_pointer_cast(d_input.data()),
    thrust::raw_pointer_cast(d_output.data()),
    begin_bit,
    end_bit, 
    striped);

  thrust::host_vector<type> h_reference =
    compute_reference(d_input, is_descending, begin_bit, end_bit);

  INFO( "striped = " << striped );
  REQUIRE( h_reference == d_output );
}

CUB_TEST("Block radix sort can sort keys in descending order",
         "[radix][sort][block]",
         types,
         no_value_types,
         items_per_thread,
         threads_in_block,
         radix_bits,
         memoize,
         algorithm,
         shmem_config)
{
  using params = params_t<TestType>;
  using type = typename params::key_type;

  thrust::device_vector<type> d_output(params::tile_size);
  thrust::device_vector<type> d_input(params::tile_size);
  c2h::gen(CUB_SEED(2), d_input);

  const int key_size  = static_cast<int>(sizeof(type) * 8);
  const int begin_bit = GENERATE_COPY(take(2, random(0, key_size)));
  const int end_bit   = GENERATE_COPY(take(2, random(begin_bit, key_size)));
  const bool striped  = GENERATE_COPY(false, true);

  const bool is_descending = true;

  block_radix_sort<params::items_per_thread,
                   params::threads_in_block,
                   params::radix_bits,
                   params::memoize,
                   params::algorithm,
                   params::shmem_config>(
    descending_sort_op_t{},
    thrust::raw_pointer_cast(d_input.data()),
    thrust::raw_pointer_cast(d_output.data()),
    begin_bit,
    end_bit,
    striped);

  thrust::host_vector<type> h_reference =
    compute_reference(d_input, is_descending, begin_bit, end_bit);

  REQUIRE( h_reference == d_output );
}

CUB_TEST("Block radix sort can sort pairs",
         "[radix][sort][block]",
         key_types,
         no_value_types,
         items_per_thread,
         threads_in_block,
         radix_bits,
         memoize,
         algorithm,
         shmem_config_4)
{
  using params = params_t<TestType>;
  using key_type = typename params::key_type;
  using value_type = key_type; 

  thrust::device_vector<key_type> d_output_keys(params::tile_size);
  thrust::device_vector<value_type> d_output_values(params::tile_size);
  thrust::device_vector<key_type> d_input_keys(params::tile_size);
  thrust::device_vector<value_type> d_input_values(params::tile_size);
  c2h::gen(CUB_SEED(2), d_input_keys);
  c2h::gen(CUB_SEED(2), d_input_values);

  const int key_size  = static_cast<int>(sizeof(key_type) * 8);
  const int begin_bit = GENERATE_COPY(take(2, random(0, key_size)));
  const int end_bit   = GENERATE_COPY(take(2, random(begin_bit, key_size)));
  const bool striped  = GENERATE_COPY(false, true);

  const bool is_descending = false;

  block_radix_sort<params::items_per_thread,
                   params::threads_in_block,
                   params::radix_bits,
                   params::memoize,
                   params::algorithm,
                   params::shmem_config>(
    sort_pairs_op_t{},
    thrust::raw_pointer_cast(d_input_keys.data()),
    thrust::raw_pointer_cast(d_input_values.data()),
    thrust::raw_pointer_cast(d_output_keys.data()),
    thrust::raw_pointer_cast(d_output_values.data()),
    begin_bit,
    end_bit,
    striped);

  std::pair<thrust::host_vector<key_type>, thrust::host_vector<value_type>>
    h_reference = compute_reference(d_input_keys,
                                    d_input_values,
                                    is_descending,
                                    begin_bit,
                                    end_bit);

  REQUIRE( h_reference.first == d_output_keys );
  REQUIRE( h_reference.second == d_output_values );
}

CUB_TEST("Block radix sort can sort pairs in descending order",
         "[radix][sort][block]",
         key_types,
         no_value_types,
         items_per_thread,
         threads_in_block,
         radix_bits,
         memoize,
         algorithm,
         shmem_config_4)
{
  using params = params_t<TestType>;
  using key_type = typename params::key_type;
  using value_type = key_type; 

  thrust::device_vector<key_type> d_output_keys(params::tile_size);
  thrust::device_vector<value_type> d_output_values(params::tile_size);
  thrust::device_vector<key_type> d_input_keys(params::tile_size);
  thrust::device_vector<value_type> d_input_values(params::tile_size);
  c2h::gen(CUB_SEED(2), d_input_keys);
  c2h::gen(CUB_SEED(2), d_input_values);

  const int key_size  = static_cast<int>(sizeof(key_type) * 8);
  const int begin_bit = GENERATE_COPY(take(2, random(0, key_size)));
  const int end_bit   = GENERATE_COPY(take(2, random(begin_bit, key_size)));
  const bool striped  = GENERATE_COPY(false, true);

  const bool is_descending = true;

  block_radix_sort<params::items_per_thread,
                   params::threads_in_block,
                   params::radix_bits,
                   params::memoize,
                   params::algorithm,
                   params::shmem_config>(
    descending_sort_pairs_op_t{},
    thrust::raw_pointer_cast(d_input_keys.data()),
    thrust::raw_pointer_cast(d_input_values.data()),
    thrust::raw_pointer_cast(d_output_keys.data()),
    thrust::raw_pointer_cast(d_output_values.data()),
    begin_bit,
    end_bit,
    striped);

  std::pair<thrust::host_vector<key_type>, thrust::host_vector<value_type>>
    h_reference = compute_reference(d_input_keys,
                                    d_input_values,
                                    is_descending,
                                    begin_bit,
                                    end_bit);

  REQUIRE( h_reference.first == d_output_keys );
  REQUIRE( h_reference.second == d_output_values );
}

CUB_TEST("Block radix sort can sort mixed pairs",
         "[radix][sort][block]",
         key_types,
         value_types,
         items_per_thread,
         threads_in_block,
         radix_bits,
         memoize,
         algorithm,
         shmem_config_4)
{
  using params = params_t<TestType>;
  using key_type = typename params::key_type;
  using value_type = typename params::value_type; 

  thrust::device_vector<key_type> d_output_keys(params::tile_size);
  thrust::device_vector<value_type> d_output_values(params::tile_size);
  thrust::device_vector<key_type> d_input_keys(params::tile_size);
  thrust::device_vector<value_type> d_input_values(params::tile_size);
  c2h::gen(CUB_SEED(2), d_input_keys);
  c2h::gen(CUB_SEED(2), d_input_values);

  const int key_size  = static_cast<int>(sizeof(key_type) * 8);
  const int begin_bit = GENERATE_COPY(take(2, random(0, key_size)));
  const int end_bit   = GENERATE_COPY(take(2, random(begin_bit, key_size)));
  const bool striped  = GENERATE_COPY(false, true);

  const bool is_descending = false;

  block_radix_sort<params::items_per_thread,
                   params::threads_in_block,
                   params::radix_bits,
                   params::memoize,
                   params::algorithm,
                   params::shmem_config>(
    sort_pairs_op_t{},
    thrust::raw_pointer_cast(d_input_keys.data()),
    thrust::raw_pointer_cast(d_input_values.data()),
    thrust::raw_pointer_cast(d_output_keys.data()),
    thrust::raw_pointer_cast(d_output_values.data()),
    begin_bit,
    end_bit,
    striped);

  std::pair<thrust::host_vector<key_type>, thrust::host_vector<value_type>>
    h_reference = compute_reference(d_input_keys,
                                    d_input_values,
                                    is_descending,
                                    begin_bit,
                                    end_bit);

  REQUIRE( h_reference.first == d_output_keys );
  REQUIRE( h_reference.second == d_output_values );
}

CUB_TEST("Block radix sort can sort mixed pairs in descending order",
         "[radix][sort][block]",
         key_types,
         value_types,
         items_per_thread,
         threads_in_block,
         radix_bits,
         memoize,
         algorithm,
         shmem_config_4)
{
  using params = params_t<TestType>;
  using key_type = typename params::key_type;
  using value_type = typename params::value_type; 

  thrust::device_vector<key_type> d_output_keys(params::tile_size);
  thrust::device_vector<value_type> d_output_values(params::tile_size);
  thrust::device_vector<key_type> d_input_keys(params::tile_size);
  thrust::device_vector<value_type> d_input_values(params::tile_size);
  c2h::gen(CUB_SEED(2), d_input_keys);
  c2h::gen(CUB_SEED(2), d_input_values);

  const int key_size  = static_cast<int>(sizeof(key_type) * 8);
  const int begin_bit = GENERATE_COPY(take(2, random(0, key_size)));
  const int end_bit   = GENERATE_COPY(take(2, random(begin_bit, key_size)));
  const bool striped  = GENERATE_COPY(false, true);

  const bool is_descending = true;

  block_radix_sort<params::items_per_thread,
                   params::threads_in_block,
                   params::radix_bits,
                   params::memoize,
                   params::algorithm,
                   params::shmem_config>(
    descending_sort_pairs_op_t{},
    thrust::raw_pointer_cast(d_input_keys.data()),
    thrust::raw_pointer_cast(d_input_values.data()),
    thrust::raw_pointer_cast(d_output_keys.data()),
    thrust::raw_pointer_cast(d_output_values.data()),
    begin_bit,
    end_bit,
    striped);

  std::pair<thrust::host_vector<key_type>, thrust::host_vector<value_type>>
    h_reference = compute_reference(d_input_keys,
                                    d_input_values,
                                    is_descending,
                                    begin_bit,
                                    end_bit);

  REQUIRE( h_reference.first == d_output_keys );
  REQUIRE( h_reference.second == d_output_values );
}

