
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloWorld() {
    printf("Hello, World from thread %d!\n", threadIdx.x);
}

int main() {
    helloWorld<<<1, 10>>>();
    hipDeviceSynchronize();
    return 0;
}
